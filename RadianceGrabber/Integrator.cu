#include <chrono>
#include <ratio>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

#include "Integrator.h"
#include "ColorTarget.h"
#include "Aggregate.h"
#include "Marshal.cuh"
#include "DeviceConfig.h"
#include "Util.h"

namespace RadGrabber
{
	/*
		TODO:: MLT ����
	*/
	/*
		4. russian roulette ���� �߰��� ���߱� ó��?
			���� ���Ŀ�
		5. Subsurface Scattering, Transmission ������ ���߿� ����
	*/

}
